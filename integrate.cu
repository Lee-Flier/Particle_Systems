#include "hip/hip_runtime.h"
// Integrate Algorithm - An important algorithm of the Particle System API with CUDA in OpenGL.
//
// Copyright 2013 by Lee Flier

/* Use the Cuda OpenGL bindings with the runtime API.
  * Device code.
  */

#include <math.h>
#include "hip/hip_vector_types.h"
#include "hip/hip_math_constants.h"
// CUDA and CUBLAS functions

#include "hip/hip_vector_types.h"
#include "vector_functions.h"
#include "thrust/device_ptr.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "..\Particle\pVec.h"
#include "ParticleGroup.h"
#include "Particle.h"
using PAPI::pVec;
using PAPI::ParticleList;
using PAPI::Particle_t;

__constant__ float globalDamping;

//欧拉积分法
struct integrate_functor_Euler
{
    float deltaTime;

    __host__ __device__
    integrate_functor_Euler(float delta_time) : deltaTime(delta_time) {}

    __host__ __device__
    template <typename Particle_t>
    __device__
    void operator()(Particle_t &m)
    {
     // new position = old position + velocity * deltaTime
     m.vel += m.gravity * deltaTime;
     m.vel *= globalDamping;
  	  m.pos += m.vel * deltaTime;
  	  m.age +=deltaTime;
    }
};

//欧拉积分法
float cudatime=0; int n=0;
extern "C" void integrateSystem_Euler(ParticleList::iterator ibegin,ParticleList::iterator iend,float deltaTime)
{  
  unsigned int numParticles = (unsigned int)(iend - ibegin);
  Particle_t &host_particle = (*ibegin);
  Particle_t* dev_particle;
  hipMalloc((void**)&dev_particle,numParticles*sizeof(Particle_t));
  hipMemcpy(dev_particle,&host_particle,numParticles*sizeof(Particle_t),hipMemcpyHostToDevice);
  thrust::device_ptr<Particle_t> d_particle(dev_particle);
  
  hipEvent_t start, stop;
  float time=0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  if(n>=200 && n<400) hipEventRecord(start,0);
  
  thrust::for_each(d_particle,d_particle+numParticles,integrate_functor_Euler(deltaTime));
  
  if (n<200 && n>=0)
  {
  	n++;
  }
  else if (n>=200 && n<400)
  {
  	hipEventRecord(stop,0);
  	hipEventSynchronize(stop);
  	hipEventElapsedTime(&time,start,stop);
  	cudatime+=time;
  	hipEventDestroy(start);
  	hipEventDestroy(stop);
  	n++;
  }
  else if(n==400)
  {
  	std::cout<<"粒子数量为"<<numParticles<<"时，CUDA device端欧拉积分算法平均耗时："<<cudatime/(n-200)<<"ms"<<std::endl;
  	n=-1;
  }

  hipMemcpy(&host_particle,dev_particle,numParticles*sizeof(Particle_t),hipMemcpyDeviceToHost);
  hipFree(dev_particle);//别忘了释放显存
}

//以下代码暂时无法解决数据对齐，暂且搁置
/*struct integrate_functor
{
    float deltaTime;

    __host__ __device__
    integrate_functor(float delta_time) : deltaTime(delta_time) {}

    template <typename Tuple>
    __device__
    void operator()(Tuple t)
    {
        //volatile float4 posData = thrust::get<0>(t);
        //volatile float4 velData = thrust::get<1>(t);
    
        // new position = old position + velocity * deltaTime
        thrust::get<0>(t) += thrust::get<1>(t) * deltaTime;

        // store new position and velocity
        //thrust::get<0>(t) = make_float4(pos, posData.w);
        //thrust::get<1>(t) = make_float4(vel, velData.w);
    }
};

 void integrateSystem(ParticleList::iterator ibegin,ParticleList::iterator iend,float deltaTime)
{
  hipEvent_t start, stop;
  float time=0;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  unsigned int numParticles = (unsigned int)(iend - ibegin);
  Particle_t &host_particle = (*ibegin);
  Particle_t *dev_particle;
  hipMalloc((void**)&dev_particle,numParticles*sizeof(Particle_t));
  hipMemcpy(dev_particle,&host_particle,numParticles*sizeof(Particle_t),hipMemcpyHostToDevice);
  
  float3 dev_pos = make_float3(dev_particle.);

  thrust::device_ptr<float3> d_pos3((float3 *)dev_pos);
    thrust::device_ptr<float3> d_vel3((float3 *)dev_vel);

    hipEventRecord(start,0);
  thrust::for_each(
        thrust::make_zip_iterator(thrust::make_tuple(d_pos3, d_vel3)),
        thrust::make_zip_iterator(thrust::make_tuple(d_pos3+numParticles, d_vel3+numParticles)),
        integrate_functor(deltaTime));
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time,start,stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  printf("GPU实现的并行欧拉积分算法消耗时间: %f ms\n",time);
}*/
